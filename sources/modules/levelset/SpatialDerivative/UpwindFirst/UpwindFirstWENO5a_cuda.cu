#include "hip/hip_runtime.h"
// CUDA runtime
#include <hip/hip_runtime.h>

#include <typedef.hpp>
#include <cuda_macro.hpp>
#include <Core/UVec.hpp>
#include <Core/CudaStream.hpp>
#include "UpwindFirstWENO5a_cuda.hpp"

#if defined(WITH_GPU)
__global__ static 
void kernel_dim0_EpsilonCalculationMethod_Constant(
	FLOAT_TYPE* dst_deriv_l_ptr,
	FLOAT_TYPE* dst_deriv_r_ptr,
	const FLOAT_TYPE* DD0_0_ptr,
	const FLOAT_TYPE* dL0_ptr,
	const FLOAT_TYPE* dL1_ptr,
	const FLOAT_TYPE* dL2_ptr,
	const FLOAT_TYPE* dR0_ptr,
	const FLOAT_TYPE* dR1_ptr,
	const FLOAT_TYPE* dR2_ptr,
	const FLOAT_TYPE weightL0,
	const FLOAT_TYPE weightL1,
	const FLOAT_TYPE weightL2,
	const FLOAT_TYPE weightR0,
	const FLOAT_TYPE weightR1,
	const FLOAT_TYPE weightR2,
	const size_t num_of_slices,
	const size_t loop_length,
	const size_t src_target_dimension_loop_size,
	const size_t first_dimension_loop_size,
	const size_t slice_length,
	const size_t thread_length_z,
	const size_t thread_length_y,
	const size_t thread_length_x
) {
	kernel_dim0_EpsilonCalculationMethod_Constant_inline(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, src_target_dimension_loop_size, first_dimension_loop_size, slice_length,
		thread_length_z, thread_length_y, thread_length_x,
		blockIdx.y, blockIdx.x, 
		blockDim.y, blockDim.x,
		threadIdx.z, threadIdx.y, threadIdx.x);
}

__global__ static
void kernel_dim0_EpsilonCalculationMethod_maxOverNeighbor(
	FLOAT_TYPE* dst_deriv_l_ptr,
	FLOAT_TYPE* dst_deriv_r_ptr,
	const FLOAT_TYPE* DD0_0_ptr,
	const FLOAT_TYPE* dL0_ptr,
	const FLOAT_TYPE* dL1_ptr,
	const FLOAT_TYPE* dL2_ptr,
	const FLOAT_TYPE* dR0_ptr,
	const FLOAT_TYPE* dR1_ptr,
	const FLOAT_TYPE* dR2_ptr,
	const FLOAT_TYPE weightL0,
	const FLOAT_TYPE weightL1,
	const FLOAT_TYPE weightL2,
	const FLOAT_TYPE weightR0,
	const FLOAT_TYPE weightR1,
	const FLOAT_TYPE weightR2,
	const size_t num_of_slices,
	const size_t loop_length,
	const size_t src_target_dimension_loop_size,
	const size_t first_dimension_loop_size,
	const size_t slice_length,
	const size_t thread_length_z,
	const size_t thread_length_y,
	const size_t thread_length_x
) {
	kernel_dim0_EpsilonCalculationMethod_maxOverNeighbor_inline(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, src_target_dimension_loop_size, first_dimension_loop_size,  slice_length,
		thread_length_z, thread_length_y, thread_length_x,
		blockIdx.y, blockIdx.x, 
		blockDim.y, blockDim.x,
		threadIdx.z, threadIdx.y, threadIdx.x);
}


void UpwindFirstWENO5a_execute_dim0_cuda (
	FLOAT_TYPE* dst_deriv_l_ptr,
	FLOAT_TYPE* dst_deriv_r_ptr,
	const FLOAT_TYPE* DD0_ptr,
	const FLOAT_TYPE* dL0_ptr,
	const FLOAT_TYPE* dL1_ptr,
	const FLOAT_TYPE* dL2_ptr,
	const FLOAT_TYPE* dR0_ptr,
	const FLOAT_TYPE* dR1_ptr,
	const FLOAT_TYPE* dR2_ptr,
	const FLOAT_TYPE weightL0,
	const FLOAT_TYPE weightL1,
	const FLOAT_TYPE weightL2,
	const FLOAT_TYPE weightR0,
	const FLOAT_TYPE weightR1,
	const FLOAT_TYPE weightR2,
	const size_t num_of_slices,
	const size_t loop_length,
	const size_t src_target_dimension_loop_size,
	const size_t first_dimension_loop_size,
	const size_t slice_length,
	const levelset::EpsilonCalculationMethod_Type epsilonCalculationMethod_Type,
	beacls::CudaStream* cudaStream
) {
	size_t num_of_threads_z;
	size_t thread_length_z;
	size_t num_of_threads_y;
	size_t num_of_blocks_y;
	size_t thread_length_y;
	size_t num_of_threads_x;
	size_t num_of_blocks_x;
	size_t thread_length_x;
	get_cuda_thread_size<size_t>(
		thread_length_z, thread_length_y, thread_length_x,
		num_of_threads_z, num_of_threads_y, num_of_threads_x,
		num_of_blocks_y, num_of_blocks_x,
		num_of_slices, loop_length, first_dimension_loop_size,
		1, 1, 8, max_num_of_threads
		);
	dim3 num_of_blocks(num_of_blocks_x, num_of_blocks_y);
	dim3 num_of_threads(num_of_threads_x, num_of_threads_y, num_of_threads_z);

	hipStream_t stream = cudaStream->get_stream();
	switch (epsilonCalculationMethod_Type) {
	case levelset::EpsilonCalculationMethod_Invalid:
	default:
//		printf("Unknown epsilonCalculationMethod %d\n", epsilonCalculationMethod_Type);
		return;
	case levelset::EpsilonCalculationMethod_Constant:
		kernel_dim0_EpsilonCalculationMethod_Constant<<<num_of_blocks,num_of_threads, 0, stream>>>(
			dst_deriv_l_ptr, dst_deriv_r_ptr,
			DD0_ptr,
			dL0_ptr, dL1_ptr, dL2_ptr, dR0_ptr, dR1_ptr, dR2_ptr,
			weightL0, weightL1, weightL2, weightR0, weightR1, weightR2,
			num_of_slices, loop_length, src_target_dimension_loop_size, first_dimension_loop_size, slice_length,
			thread_length_z, thread_length_y, thread_length_x
			);
		break;
	case levelset::EpsilonCalculationMethod_maxOverGrid:
//		printf("epsilonCalculationMethod %d is not supported yet\n", epsilonCalculationMethod_Type);
		return;
	case levelset::EpsilonCalculationMethod_maxOverNeighbor:
		kernel_dim0_EpsilonCalculationMethod_maxOverNeighbor<<<num_of_blocks,num_of_threads, 0, stream>>>(
			dst_deriv_l_ptr, dst_deriv_r_ptr,
			DD0_ptr,
			dL0_ptr, dL1_ptr, dL2_ptr, dR0_ptr, dR1_ptr, dR2_ptr,
			weightL0, weightL1, weightL2, weightR0, weightR1, weightR2,
			num_of_slices, loop_length, src_target_dimension_loop_size, first_dimension_loop_size, slice_length,
			thread_length_z, thread_length_y, thread_length_x
			);
		break;
	}
}

__global__ static 
void kernel_dim1_EpsilonCalculationMethod_Constant(
	FLOAT_TYPE* dst_deriv_l_ptr,
	FLOAT_TYPE* dst_deriv_r_ptr,
	const FLOAT_TYPE* DD0_ptr,
	const FLOAT_TYPE* dL0_ptr,
	const FLOAT_TYPE* dL1_ptr,
	const FLOAT_TYPE* dL2_ptr,
	const FLOAT_TYPE* dR0_ptr,
	const FLOAT_TYPE* dR1_ptr,
	const FLOAT_TYPE* dR2_ptr,
	const FLOAT_TYPE weightL0,
	const FLOAT_TYPE weightL1,
	const FLOAT_TYPE weightL2,
	const FLOAT_TYPE weightR0,
	const FLOAT_TYPE weightR1,
	const FLOAT_TYPE weightR2,
	const size_t num_of_slices,
	const size_t loop_length,
	const size_t first_dimension_loop_size,
	const size_t slice_length,
	const size_t DD0_slice_size,
	const size_t thread_length_z,
	const size_t thread_length_y,
	const size_t thread_length_x
) {
	kernel_dim1_EpsilonCalculationMethod_Constant_inline(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, first_dimension_loop_size, slice_length, 
		DD0_slice_size,
		thread_length_z, thread_length_y, thread_length_x,
		blockIdx.y, blockIdx.x, 
		blockDim.y, blockDim.x,
		threadIdx.z, threadIdx.y, threadIdx.x);
}

__global__ static
void kernel_dim1_EpsilonCalculationMethod_maxOverNeighbor(
	FLOAT_TYPE* dst_deriv_l_ptr,
	FLOAT_TYPE* dst_deriv_r_ptr,
	const FLOAT_TYPE* DD0_ptr,
	const FLOAT_TYPE* dL0_ptr,
	const FLOAT_TYPE* dL1_ptr,
	const FLOAT_TYPE* dL2_ptr,
	const FLOAT_TYPE* dR0_ptr,
	const FLOAT_TYPE* dR1_ptr,
	const FLOAT_TYPE* dR2_ptr,
	const FLOAT_TYPE weightL0,
	const FLOAT_TYPE weightL1,
	const FLOAT_TYPE weightL2,
	const FLOAT_TYPE weightR0,
	const FLOAT_TYPE weightR1,
	const FLOAT_TYPE weightR2,
	const size_t num_of_slices,
	const size_t loop_length,
	const size_t first_dimension_loop_size,
	const size_t slice_length,
	const size_t DD0_slice_size,
	const size_t thread_length_z,
	const size_t thread_length_y,
	const size_t thread_length_x
) {
	kernel_dim1_EpsilonCalculationMethod_maxOverNeighbor_inline(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, first_dimension_loop_size, slice_length, 
		DD0_slice_size,
		thread_length_z, thread_length_y, thread_length_x,
		blockIdx.y, blockIdx.x, 
		blockDim.y, blockDim.x,
		threadIdx.z, threadIdx.y, threadIdx.x);
}


void UpwindFirstWENO5a_execute_dim1_cuda (
	FLOAT_TYPE* dst_deriv_l_ptr,
	FLOAT_TYPE* dst_deriv_r_ptr,
	const FLOAT_TYPE* DD0_ptr,
	const FLOAT_TYPE* dL0_ptr,
	const FLOAT_TYPE* dL1_ptr,
	const FLOAT_TYPE* dL2_ptr,
	const FLOAT_TYPE* dR0_ptr,
	const FLOAT_TYPE* dR1_ptr,
	const FLOAT_TYPE* dR2_ptr,
	const FLOAT_TYPE weightL0,
	const FLOAT_TYPE weightL1,
	const FLOAT_TYPE weightL2,
	const FLOAT_TYPE weightR0,
	const FLOAT_TYPE weightR1,
	const FLOAT_TYPE weightR2,
	const size_t num_of_slices,
	const size_t loop_length,
	const size_t first_dimension_loop_size,
	const size_t slice_length,
	const size_t DD0_slice_size,
	const levelset::EpsilonCalculationMethod_Type epsilonCalculationMethod_Type,
	beacls::CudaStream* cudaStream
) {
	size_t num_of_threads_z;
	size_t thread_length_z;
	size_t num_of_threads_y;
	size_t num_of_blocks_y;
	size_t thread_length_y;
	size_t num_of_threads_x;
	size_t num_of_blocks_x;
	size_t thread_length_x;
	get_cuda_thread_size<size_t>(
		thread_length_z, thread_length_y, thread_length_x,
		num_of_threads_z, num_of_threads_y, num_of_threads_x,
		num_of_blocks_y, num_of_blocks_x,
		num_of_slices, loop_length, first_dimension_loop_size,
		1, 8, 1, max_num_of_threads
		);
	dim3 num_of_blocks(num_of_blocks_x, num_of_blocks_y);
	dim3 num_of_threads(num_of_threads_x, num_of_threads_y, num_of_threads_z);
	hipStream_t stream = cudaStream->get_stream();

	switch (epsilonCalculationMethod_Type) {
	case levelset::EpsilonCalculationMethod_Invalid:
	default:
//		printf("Unknown epsilonCalculationMethod %d\n", epsilonCalculationMethod_Type);
		return;
	case levelset::EpsilonCalculationMethod_Constant:
		kernel_dim1_EpsilonCalculationMethod_Constant<<<num_of_blocks,num_of_threads, 0, stream>>>(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, first_dimension_loop_size, slice_length, 
		DD0_slice_size,
		thread_length_z, thread_length_y, thread_length_x
		);
		break;
	case levelset::EpsilonCalculationMethod_maxOverGrid:
//		printf("epsilonCalculationMethod %d is not supported yet\n", epsilonCalculationMethod_Type);
		return;
	case levelset::EpsilonCalculationMethod_maxOverNeighbor:
		kernel_dim1_EpsilonCalculationMethod_maxOverNeighbor<<<num_of_blocks,num_of_threads, 0, stream>>>(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, first_dimension_loop_size, slice_length, 
		DD0_slice_size,
		thread_length_z, thread_length_y, thread_length_x
				);
		break;
	}
}

__global__ static 
void kernel_dimLET2_EpsilonCalculationMethod_Constant(
	FLOAT_TYPE* dst_deriv_l_ptr,
	FLOAT_TYPE* dst_deriv_r_ptr,
	const FLOAT_TYPE* DD0_0_ptr,
	const FLOAT_TYPE* DD1_0_ptr,
	const FLOAT_TYPE* DD2_0_ptr,
	const FLOAT_TYPE* DD3_0_ptr,
	const FLOAT_TYPE* DD4_0_ptr,
	const FLOAT_TYPE* DD5_0_ptr,
	const FLOAT_TYPE* dL0_ptr,
	const FLOAT_TYPE* dL1_ptr,
	const FLOAT_TYPE* dL2_ptr,
	const FLOAT_TYPE* dR0_ptr,
	const FLOAT_TYPE* dR1_ptr,
	const FLOAT_TYPE* dR2_ptr,
	const FLOAT_TYPE weightL0,
	const FLOAT_TYPE weightL1,
	const FLOAT_TYPE weightL2,
	const FLOAT_TYPE weightR0,
	const FLOAT_TYPE weightR1,
	const FLOAT_TYPE weightR2,
	const size_t num_of_slices,
	const size_t loop_length,
	const size_t first_dimension_loop_size,
	const size_t slice_length,
	const size_t thread_length_z,
	const size_t thread_length_y,
	const size_t thread_length_x
) {
	kernel_dimLET2_EpsilonCalculationMethod_Constant_inline(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_0_ptr,DD1_0_ptr,DD2_0_ptr, DD3_0_ptr, DD4_0_ptr, DD5_0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, first_dimension_loop_size, slice_length, 
		thread_length_z, thread_length_y, thread_length_x,
		blockIdx.y, blockIdx.x, 
		blockDim.y, blockDim.x,
		threadIdx.z, threadIdx.y, threadIdx.x);
}

__global__ static
void kernel_dimLET2_EpsilonCalculationMethod_maxOverNeighbor(
	FLOAT_TYPE* dst_deriv_l_ptr,
	FLOAT_TYPE* dst_deriv_r_ptr,
	const FLOAT_TYPE* DD0_0_ptr,
	const FLOAT_TYPE* DD1_0_ptr,
	const FLOAT_TYPE* DD2_0_ptr,
	const FLOAT_TYPE* DD3_0_ptr,
	const FLOAT_TYPE* DD4_0_ptr,
	const FLOAT_TYPE* DD5_0_ptr,
	const FLOAT_TYPE* dL0_ptr,
	const FLOAT_TYPE* dL1_ptr,
	const FLOAT_TYPE* dL2_ptr,
	const FLOAT_TYPE* dR0_ptr,
	const FLOAT_TYPE* dR1_ptr,
	const FLOAT_TYPE* dR2_ptr,
	const FLOAT_TYPE weightL0,
	const FLOAT_TYPE weightL1,
	const FLOAT_TYPE weightL2,
	const FLOAT_TYPE weightR0,
	const FLOAT_TYPE weightR1,
	const FLOAT_TYPE weightR2,
	const size_t num_of_slices,
	const size_t loop_length,
	const size_t first_dimension_loop_size,
	const size_t slice_length,
	const size_t thread_length_z,
	const size_t thread_length_y,
	const size_t thread_length_x
) {
	kernel_dimLET2_EpsilonCalculationMethod_maxOverNeighbor_inline(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_0_ptr,DD1_0_ptr,DD2_0_ptr, DD3_0_ptr, DD4_0_ptr, DD5_0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, first_dimension_loop_size, slice_length,
		thread_length_z, thread_length_y, thread_length_x,
		blockIdx.y, blockIdx.x, 
		blockDim.y, blockDim.x,
		threadIdx.z, threadIdx.y, threadIdx.x);
}


void UpwindFirstWENO5a_execute_dimLET2_cuda (
	FLOAT_TYPE* dst_deriv_l_ptr,
	FLOAT_TYPE* dst_deriv_r_ptr,
	const FLOAT_TYPE* DD0_0_ptr,
	const FLOAT_TYPE* DD1_0_ptr,
	const FLOAT_TYPE* DD2_0_ptr,
	const FLOAT_TYPE* DD3_0_ptr,
	const FLOAT_TYPE* DD4_0_ptr,
	const FLOAT_TYPE* DD5_0_ptr,
	const FLOAT_TYPE* dL0_ptr,
	const FLOAT_TYPE* dL1_ptr,
	const FLOAT_TYPE* dL2_ptr,
	const FLOAT_TYPE* dR0_ptr,
	const FLOAT_TYPE* dR1_ptr,
	const FLOAT_TYPE* dR2_ptr,
	const FLOAT_TYPE weightL0,
	const FLOAT_TYPE weightL1,
	const FLOAT_TYPE weightL2,
	const FLOAT_TYPE weightR0,
	const FLOAT_TYPE weightR1,
	const FLOAT_TYPE weightR2,
	const size_t num_of_slices,
	const size_t loop_length,
	const size_t first_dimension_loop_size,
	const size_t slice_length,
	const levelset::EpsilonCalculationMethod_Type epsilonCalculationMethod_Type,
	beacls::CudaStream* cudaStream
) {
	size_t num_of_threads_z;
	size_t thread_length_z;
	size_t num_of_threads_y;
	size_t num_of_blocks_y;
	size_t thread_length_y;
	size_t num_of_threads_x;
	size_t num_of_blocks_x;
	size_t thread_length_x;
	get_cuda_thread_size<size_t>(
		thread_length_z, thread_length_y, thread_length_x,
		num_of_threads_z, num_of_threads_y, num_of_threads_x,
		num_of_blocks_y, num_of_blocks_x,
		num_of_slices, loop_length, first_dimension_loop_size,
		1, 1, 1, max_num_of_threads
		);
	dim3 num_of_blocks(num_of_blocks_x, num_of_blocks_y);
	dim3 num_of_threads(num_of_threads_x, num_of_threads_y, num_of_threads_z);

	hipStream_t stream = cudaStream->get_stream();
	switch (epsilonCalculationMethod_Type) {
	case levelset::EpsilonCalculationMethod_Invalid:
	default:
//		printf("Unknown epsilonCalculationMethod %d\n", epsilonCalculationMethod_Type);
		return;
	case levelset::EpsilonCalculationMethod_Constant:
		kernel_dimLET2_EpsilonCalculationMethod_Constant<<<num_of_blocks,num_of_threads, 0, stream>>>(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_0_ptr,DD1_0_ptr,DD2_0_ptr, DD3_0_ptr, DD4_0_ptr, DD5_0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, first_dimension_loop_size, slice_length, 
		thread_length_z, thread_length_y, thread_length_x
		);
		break;
	case levelset::EpsilonCalculationMethod_maxOverGrid:
//		printf("epsilonCalculationMethod %d is not supported yet\n", epsilonCalculationMethod_Type);
		return;
	case levelset::EpsilonCalculationMethod_maxOverNeighbor:
		kernel_dimLET2_EpsilonCalculationMethod_maxOverNeighbor<<<num_of_blocks,num_of_threads, 0, stream>>>(
		dst_deriv_l_ptr, dst_deriv_r_ptr,
		DD0_0_ptr,DD1_0_ptr,DD2_0_ptr, DD3_0_ptr, DD4_0_ptr, DD5_0_ptr,
		dL0_ptr, dL1_ptr, dL2_ptr,dR0_ptr, dR1_ptr, dR2_ptr,
		weightL0, weightL1, weightL2, weightR0, weightR1, weightR2, 
		num_of_slices, loop_length, first_dimension_loop_size, slice_length, 
		thread_length_z, thread_length_y, thread_length_x
		);
		break;
	}
}
#endif /* defined(WITH_GPU) */
